#include "hip/hip_runtime.h"
#include <string.h>
#include "Motion.h"
#include <cmath>
#include <stdlib.h>
#include <stdio.h>


#define Clip3(a, b, c) ((c < a) ? a : ((c > b) ? b : c))
#define Clip1(x)       Clip3(0, 255, x)


__global__  void Motion_Estimation_tool_cuda(int* d_lastFrameY, int* d_currentFrameY, int* d_resultFrameY, int* d_interpolation_block, int* d_temp_interpolation_block, int block_size, int search_region_R, int im_height, int im_width)
{
	int SAD_min = 1000000;
	int current_SAD;
	int SAD_min_x = 0, SAD_min_y = 0;


	// the top-left corner of this block on current frame and search block on last frame
	int search_block_x, search_block_y;
	int search_x, search_y, refer_x, refer_y;
	int delta_x, delta_y;

	const int block_x = threadIdx.x * block_size;
	const int block_y = threadIdx.y * block_size;

	bool inter_better_flag = false;



#pragma region Homework4
	for (delta_x = -search_region_R; delta_x <= search_region_R; delta_x++)
	{
		for (delta_y = -search_region_R; delta_y <= search_region_R; delta_y++)
		{
			search_block_x = block_x + delta_x;
			search_block_y = block_y + delta_y;
			if ((search_block_x >= 0) && (search_block_x <= im_height - block_size) && (search_block_y >= 0) && (search_block_y <= im_width - block_size))
			{
				// block position on lastFrameY
				current_SAD = 0;
				for (int m = 0; m < block_size; m++)
				{
					for (int n = 0; n < block_size; n++)
					{
						search_x = search_block_x + m;
						search_y = search_block_y + n;
						refer_x = block_x + m;
						refer_y = block_y + n;
						current_SAD += fabsf(d_lastFrameY[search_x * im_width + search_y] - d_currentFrameY[refer_x * im_width + refer_y]);
					}
				}
				if (current_SAD < SAD_min)
				{
					SAD_min_x = delta_x;
					SAD_min_y = delta_y;
					SAD_min = current_SAD;
				}
			}
		}
	}
#pragma endregion




#pragma region Homework5

	int inter_SAD_min = 1000000, inter_SAD;
	
	for (int flag = 1; flag <= 16; flag++)
	{

		//motion_vector of 1/2 pixel motion estimation
		switch (flag) {
		case 1:
			delta_x = -1;
			delta_y = -1;
			break;
		case 2:
			delta_x = -1;
			delta_y = 0;
			break;
		case 3:
			delta_x = 0;
			delta_y = -1;
			break;
		case 4:
			delta_x = 0;
			delta_y = 0;
			break;
		case 5:
			delta_x = 1;
			delta_y = -1;
			break;
		case 6:
			delta_x = 1;
			delta_y = 0;
			break;
		case 7:
			delta_x = -1;
			delta_y = -1;
			break;
		case 8:
			delta_x = -1;
			delta_y = 0;
			break;
		case 9:
			delta_x = -1;
			delta_y = 1;
			break;
		case 10:
			delta_x = 0;
			delta_y = -1;
			break;
		case 11:
			delta_x = 0;
			delta_y = 0;
			break;
		case 12:
			delta_x = 0;
			delta_y = 1;
			break;
		case 13:
			delta_x = -1;
			delta_y = -1;
			break;
		case 14:
			delta_x = -1;
			delta_y = 0;
			break;
		case 15:
			delta_x = 0;
			delta_y = -1;
			break;
		case 16:
			delta_x = 0;
			delta_y = 0;
			break;
		}

		search_block_x = block_x + SAD_min_x + delta_x;
		search_block_y = block_y + SAD_min_y + delta_y;

		if ((search_block_x >= 0) && (search_block_x <= im_height - block_size) && (search_block_y >= 0) && (search_block_y <= im_width - block_size))
		{
			inter_SAD = 0;

			for (int m = 0; m < block_size; m++)
			{
				for (int n = 0; n < block_size; n++)
				{
					if (flag >= 1 && flag <= 6)  //the first kind of interpolation
					{
						d_temp_interpolation_block[m * im_width + n] = (d_lastFrameY[(search_block_x + m) * im_width + search_block_y + n] + d_lastFrameY[(search_block_x + m) * im_width + search_block_y + n + 1]) / 2;
					}
					else if (flag >= 7 && flag <= 12)
					{
						d_temp_interpolation_block[m * im_width + n] = (d_lastFrameY[(search_block_x + m) * im_width + search_block_y + n] + d_lastFrameY[(search_block_x + m + 1) * im_width + search_block_y + n]) / 2;
					}
					else if (flag >= 13 && flag <= 16)
					{
						d_temp_interpolation_block[m * im_width + n] = (d_lastFrameY[(search_block_x + m) * block_size + search_block_y + n] + d_lastFrameY[(search_block_x + m + 1) * block_size + search_block_y + n]) / 2;
					}
					refer_x = block_x + m;
					refer_y = block_y + n;
					inter_SAD += fabsf(d_temp_interpolation_block[m * im_width + n] - d_currentFrameY[refer_x * im_width + refer_y]);
				}
			}

			if (inter_SAD < inter_SAD_min)
			{
				for (int m = 1; m <= block_size; m++)
				{
					for (int n = 1; n <= block_size; n++)
					{
						d_interpolation_block[m * im_width + n] = d_temp_interpolation_block[m * im_width + n];
					}
				}
			}

		}

		if (inter_SAD_min < SAD_min)
		{
			SAD_min = inter_SAD_min;
			inter_better_flag = true;
		}
	}
#pragma endregion
	

#pragma region Update Result

	if (inter_better_flag)
	{
		for (int m = 0; m < block_size; m++)
		{
			for (int n = 0; n < block_size; n++)
			{
				d_resultFrameY[(block_x + m) * im_width + (block_y + n)] = d_interpolation_block[m * im_width + n];
			}
		}
	}
	else
	{
		for (int m = 0; m < block_size; m++)
		{
			for (int n = 0; n < block_size; n++)
			{
				d_resultFrameY[(block_x + m) * im_width + (block_y + n)] = d_lastFrameY[(block_x + SAD_min_x + m) * im_width + (block_y + SAD_min_y + n)];
			}
		}
	}

#pragma endregion

}



void Motion_Estimation_cuda(int* lastFrameY, int* currentFrameY, int* resultFrameY, int block_size, int search_region_R, int im_height, int im_width)
{
	size_t size = im_height * im_width * sizeof(int);


	// Allocate memory
	int* d_currentFrameY;
	hipMalloc(&d_currentFrameY, size);
	int* d_lastFrameY;
	hipMalloc(&d_lastFrameY, size);
	int* d_resultFrameY;
	hipMalloc(&d_resultFrameY, size);

	int* d_interpolation_block;
	hipMalloc(&d_interpolation_block, size);
	int* d_temp_interpolation_block;
	hipMalloc(&d_temp_interpolation_block, size);


	// Copy vectors from host memory to device memory
	hipMemcpy(d_currentFrameY, currentFrameY, size, hipMemcpyHostToDevice);
	hipMemcpy(d_lastFrameY, lastFrameY, size, hipMemcpyHostToDevice);
	hipMemcpy(d_resultFrameY, resultFrameY, size, hipMemcpyHostToDevice);


	// Invoke kernel
	dim3 threadsPerBlock(im_height / block_size, im_width / block_size);
	Motion_Estimation_tool_cuda << <1, threadsPerBlock >> > (d_lastFrameY, d_currentFrameY, d_resultFrameY, d_interpolation_block, d_temp_interpolation_block,  block_size, search_region_R, im_height, im_width);


	//// Copy result from device memory to host memory
	hipMemcpy(resultFrameY, d_resultFrameY, size, hipMemcpyDeviceToHost);


	// Free device memory
	hipFree(d_currentFrameY);
	hipFree(d_lastFrameY);
	hipFree(d_resultFrameY);

	hipFree(d_interpolation_block);
	hipFree(d_temp_interpolation_block);

}
